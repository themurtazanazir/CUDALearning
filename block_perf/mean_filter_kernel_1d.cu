#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAStream.h>
#include <stdio.h>


using namespace std;
__global__
void mean_filter_kernel_1d(unsigned char *output, unsigned char *input, int width, int height, int radius){
    int idx = blockDim.x*blockIdx.x + threadIdx.x;

    int row = idx/width;
    int col = idx%width;

    if (row < height && col < width){
        int val = 0;
        int count = 0;
        for (int blurRow = -radius; blurRow < radius; ++blurRow)
        {
            for (int blurCol = -radius; blurCol < radius; ++blurCol)
            {
                int i = row + blurRow;
                int j = col + blurCol;
                if (i < height && j < width && i >= 0 && j >= 0)
                {
                    val += input[width * i + j];
                    count++;
                }
            }
        }
        output[idx] = (unsigned char)(val / count);
    } 
}



torch::Tensor mean_filter_1d(torch::Tensor image, int radius)
{
    assert(image.device().type() == torch::kCUDA);
    assert(image.dtype() == torch::kByte);
    assert(radius > 0);

    const auto height = image.size(0);
    const auto width = image.size(1);

    auto result = torch::empty_like(image);

    dim3 threads_per_block(512);
    auto size = width*height;
    dim3 blocks_per_grid(ceil((float)size / threads_per_block.x));
    // cout<<blocks_per_grid.x<<" "<<blocks_per_grid.y<<endl;
    // cout<<threads_per_block.x<<" "<<threads_per_block.y<<endl;
    mean_filter_kernel_1d<<<blocks_per_grid, threads_per_block, 0, torch::cuda::getCurrentCUDAStream()>>>(
        result.data_ptr<unsigned char>(),
        image.data_ptr<unsigned char>(),
        width,
        height,
        radius);

    C10_CUDA_KERNEL_LAUNCH_CHECK();
    return result;
}